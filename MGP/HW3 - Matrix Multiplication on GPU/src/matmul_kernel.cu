#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include "matmul.h"
using namespace std;

void allocateDeviceMemory(void** M, int size)
{
  hipError_t err = hipMalloc(M, size);
  assert(err==hipSuccess);
}

void deallocateDeviceMemory(void* M)
{
  hipError_t err = hipFree(M);
  assert(err==hipSuccess);
}

void matmul_ref(const int* const matrixA, const int* const matrixB,
                int* const matrixC, const int n) {
  // You can assume matrixC is initialized with zero
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      for (int k = 0; k < n; k++)
        matrixC[i * n + j] += matrixA[i * n + k] * matrixB[k * n + j];
}

#define TILE_WIDTH 32
__global__ void MatrixMulKernel(const int* M, const int* N, int* const P, int Width) {
  __shared__ int subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ int subTileN[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  int Pvalue = 0;

  for (int m = 0; m < (Width)/(TILE_WIDTH); m++) {
    if (Row < Width && m * TILE_WIDTH + tx < Width) subTileM[ty][tx] = M[Row * Width + m * TILE_WIDTH + tx];
    else subTileM[ty][tx] = 0;
    
    if (Col < Width && m * TILE_WIDTH + ty < Width) subTileN[ty][tx] = N[(m * TILE_WIDTH + ty) * Width + Col];
    else subTileN[ty][tx] = 0;
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; k++) { 
      Pvalue += subTileM[ty][k] * subTileN[k][tx];
    }
    __syncthreads();
  }

  if (Row < Width && Col < Width) P[Row * Width + Col] = Pvalue;
}

void matmul_optimized(const int* const matrixA, const int* const matrixB,
                      int* const matrixC, const int* const d_A, const int* const d_B,  int* const d_C, const int n) {

  // TODO: Implement your CUDA code
  int* device_A = (int *)d_A;
  int* device_B = (int *)d_B;
  int size = n * n * sizeof(int);
  hipMemcpy(device_A, matrixA, size, hipMemcpyHostToDevice);
  hipMemcpy(device_B, matrixB, size, hipMemcpyHostToDevice);

  dim3 dimGrid(n/TILE_WIDTH, n/TILE_WIDTH, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);

  // copy the matrix multiplication result
  hipMemcpy(matrixC, d_C, size, hipMemcpyDeviceToHost);
  return;
}

