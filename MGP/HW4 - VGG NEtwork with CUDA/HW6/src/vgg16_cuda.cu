#include "hip/hip_runtime.h"
#include "vgg16_cuda.h"
__global__
void cuda_normalize(const uint8_t* const image, float* input) {
    // Initialize variables
    float max_int = 255.0L;
    float mean = 0.5L;
    float var = 0.5L;

    // index calculation
    int B = gridDim.x; int b = blockIdx.x;
    int C = gridDim.y; int c = blockIdx.y;
    int W = blockDim.x; int w = threadIdx.x;
    int H = blockDim.y; int h = threadIdx.y;
    int idx = b * (C * H * W) + c * (H * W) + h * (W) + w;

    // Normalize
    if (idx < B * C * W * H) {
        input[idx] = image[idx] / max_int;       // transforms.ToTensor();
        input[idx] = (input[idx] - mean) / var;  // transforms.Normalize();
    }
}

__global__
void cuda_relu(float* feature_map) {
    // index calculation
    int B = gridDim.x; int b = blockIdx.x;
    int C = gridDim.y; int c = blockIdx.y;
    int W = blockDim.x; int w = threadIdx.x;
    int H = blockDim.y; int h = threadIdx.y;
    int idx = b * (C * H * W) + c * (H * W) + h * (W) + w;

    // relu if (idx < size) 
    if (idx < B * C * W * H) {
        feature_map[idx] = feature_map[idx] > (float)0.0 ? feature_map[idx] : (float)0.0;
    }
}

__global__
void cuda_pad(float* input, float* input_padded, int P) {
    // variable
    int B = gridDim.x; int b = blockIdx.x;
    int C = gridDim.y; int c = blockIdx.y;
    int W = blockDim.x; int w = threadIdx.x;
    int H = blockDim.y; int h = threadIdx.y;

    // length changed after padding
    int H_OUT = H + 2 * P;
    int W_OUT = W + 2 * P;

    int input_base = b * (C * H * W) + c * (H * W) + h * (W) + w;
    if (input_base < B * C * W * H) {
        int output_idx = b * (C * H_OUT * W_OUT) + c * (H_OUT * W_OUT) + (h + P) * W_OUT + (w + P);
        input_padded[output_idx] = input[input_base];
    }
}

__global__
void cuda_conv(float* input, float* output, float* weight, float* bias, int IC, int K) {
    int B = gridDim.x; int b = blockIdx.x;
    int OC = gridDim.y; int oc = blockIdx.y;
    int W_OUT = blockDim.x; int w = threadIdx.x;
    int H_OUT = blockDim.y; int h = threadIdx.y;
    int W = W_OUT + (K - 1); int H = H_OUT + (K - 1);

    // convolution
    int output_idx = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
    if (output_idx < B * OC * H_OUT * W_OUT) {
        float acc = bias[oc];
        for (int ic = 0; ic < IC; ic++) {
            int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
            int kernel_base = oc * (IC * K * K) + ic * (K * K);

            for (int kh = 0; kh < K; kh++)
                for (int kw = 0; kw < K; kw++) {
                    acc += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
                }
        }
        output[output_idx] = acc;
    }
}

__global__
void cuda_conv_b5(float* input, float* output, float* weight, float* bias, int output_size, int IC, int K) {
    int B = gridDim.x; int b = blockIdx.x;
    int OC = blockDim.x; int oc = threadIdx.x;

    int W_OUT = output_size; int H_OUT = output_size;
    int W = W_OUT + (K - 1); int H = H_OUT + (K - 1);

    __shared__ float bias_shared[512];

    // convolution
    int block_id = b * (OC) + oc;
    if (block_id < B * OC) {
        for (int bi = 0; bi < 512; bi++) bias_shared[bi] = bias[bi];
        __syncthreads();

        for (int h = 0; h < H_OUT; h++)
            for (int w = 0; w < W_OUT; w++) {
                float acc = 0;
                int output_idx = b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
                for (int ic = 0; ic < IC; ic++) {
                    int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
                    int kernel_base = oc * (IC * K * K) + ic * (K * K);
                    for (int kh = 0; kh < K; kh++)
                        for (int kw = 0; kw < K; kw++) {
                            acc += input[input_base + kh * (W) + kw] * weight[kernel_base + kh * (K) + kw];
                        }
                }
                output[output_idx] = acc + bias_shared[oc];
            }
    }
}

__global__
void cuda_pool(float* input, float* output) {
    // Initilaize variable
    int B = gridDim.x; int b = blockIdx.x;
    int C = gridDim.y; int c = blockIdx.y;
    int W_OUT = blockDim.x; int w = threadIdx.x;
    int H_OUT = blockDim.y; int h = threadIdx.y;

    int scale = 2;
    int H = H_OUT * scale;
    int W = W_OUT * scale;

    // Max Pooling
    // Init values
    float max_val = -256;
    int output_index = b * (C * H_OUT * W_OUT) + c * (H_OUT * W_OUT) + h * W_OUT + w;
    if (output_index < B * C * H_OUT * W_OUT) {
        // Find maximum
        for (int sh = 0; sh < scale; sh++)
            for (int sw = 0; sw < scale; sw++) {
                float val = input[b * (C * H * W) + c * (H * W) + (2 * h + sh) * (W) + (2 * w + sw)];
                if (val - max_val > 0) {
                    max_val = val;
                }
            }
        // Set output with max value
        output[output_index] = max_val;
    }
}

__global__
void cuda_fc(float* input, float* output, float* weight, float* bias, int IC) {
    int B = gridDim.x; int b = blockIdx.x;
    int OC = blockDim.x; int oc = threadIdx.x;
    int idx = b * OC + oc;

    if (idx < B * OC) {
        float acc = bias[oc];
        for (int ic = 0; ic < IC; ic++) {
            acc += weight[oc * IC + ic] * input[b * IC + ic];
        }
        output[idx] = acc;
    }
}

void vgg16_cuda::predict(int batch) {
    //////////////////////////////////////////////////
    // Normalization
    dim3 dimBlock_norm(input_size, input_size);
    dim3 dimGrid_norm(batch, input_channel);
    cuda_normalize<<<dimGrid_norm, dimBlock_norm>>>(d_image, d_input);

    //////////BLOCK 1/////////////////////////////////
    // TODO: Implement pad
    dim3 dimBlock_pad1_1(input_size, input_size);
    dim3 dimGrid_pad1_1(batch, conv1_1_in_channel);
    cuda_pad<<<dimGrid_pad1_1, dimBlock_pad1_1>>>(d_input, d_input_padded, conv1_1_padding_size);

    // TODO: Implement conv1_1
    dim3 dimBlock_conv1_1(C1_1_size, C1_1_size);
    dim3 dimGrid_conv1_1(batch, C1_1_channel);
    cuda_conv<<<dimGrid_conv1_1, dimBlock_conv1_1>>>(d_input_padded, d_C1_1_feature_map, d_conv1_1_weight, d_conv1_1_bias, 
                                                        conv1_1_in_channel, conv1_1_kernel_size);                                           

    // TODO: Implement relu
    dim3 dimBlock_relu1_1(C1_1_size, C1_1_size);
    dim3 dimGrid_relu1_1(batch, C1_1_channel);
    cuda_relu<<<dimGrid_relu1_1, dimBlock_relu1_1>>>(d_C1_1_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad1_2(C1_1_size, C1_1_size);
    dim3 dimGrid_pad1_2(batch, C1_1_channel);
    cuda_pad<<<dimGrid_pad1_2, dimBlock_pad1_2>>>(d_C1_1_feature_map, d_C1_1_feature_map_padded, conv1_2_padding_size);

    // TODO: Implement conv1_2
    dim3 dimBlock_conv1_2(C1_2_size, C1_2_size);
    dim3 dimGrid_conv1_2(batch, C1_2_channel);
    cuda_conv<<<dimGrid_conv1_2, dimBlock_conv1_2>>>(d_C1_1_feature_map_padded, d_C1_2_feature_map, d_conv1_2_weight, d_conv1_2_bias, 
                                                        conv1_2_in_channel, conv1_2_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu1_2(C1_2_size, C1_2_size);
    dim3 dimGrid_relu1_2(batch, C1_2_channel);
    cuda_relu<<<dimGrid_relu1_2, dimBlock_relu1_2>>>(d_C1_2_feature_map);

    // TODO: Implement pool
    dim3 dimBlock_pool1(S1_size, S1_size);
    dim3 dimGrid_pool1(batch, S1_channel);
    cuda_pool<<<dimGrid_pool1, dimBlock_pool1>>>(d_C1_2_feature_map, d_S1_feature_map);    

    //////////BLOCK 2/////////////////////////////////
    // TODO: Implement pad
    dim3 dimBlock_pad2_1(S1_size, S1_size);
    dim3 dimGrid_pad2_1(batch, S1_channel);
    cuda_pad<<<dimGrid_pad2_1, dimBlock_pad2_1>>>(d_S1_feature_map, d_S1_feature_map_padded, conv2_1_padding_size);

    // TODO: Implement conv2_1
    dim3 dimBlock_conv2_1(C2_1_size, C2_1_size);
    dim3 dimGrid_conv2_1(batch, C2_1_channel);
    cuda_conv<<<dimGrid_conv2_1, dimBlock_conv2_1>>>(d_S1_feature_map_padded, d_C2_1_feature_map, d_conv2_1_weight, d_conv2_1_bias,
                                                        conv2_1_in_channel, conv2_1_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu2_1(C2_1_size, C2_1_size);
    dim3 dimGrid_relu2_1(batch, C2_1_channel);
    cuda_relu<<<dimGrid_relu2_1, dimBlock_relu2_1>>>(d_C2_1_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad2_2(C2_1_size, C2_1_size);
    dim3 dimGrid_pad2_2(batch, C2_1_channel);
    cuda_pad<<<dimGrid_pad2_2, dimBlock_pad2_2>>>(d_C2_1_feature_map, d_C2_1_feature_map_padded, conv2_2_padding_size);

    // TODO: Implement conv2_2
    dim3 dimBlock_conv2_2(C2_2_size, C2_2_size);
    dim3 dimGrid_conv2_2(batch, C2_2_channel);
    cuda_conv<<<dimGrid_conv2_2, dimBlock_conv2_2>>>(d_C2_1_feature_map_padded, d_C2_2_feature_map, d_conv2_2_weight, d_conv2_2_bias,
                                                        conv2_2_in_channel, conv2_2_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu2_2(C2_2_size, C2_2_size);
    dim3 dimGrid_relu2_2(batch, C2_2_channel);
    cuda_relu<<<dimGrid_relu2_2, dimBlock_relu2_2>>>(d_C2_2_feature_map);

    // TODO: Implement pool
    dim3 dimBlock_pool2(S2_size, S2_size);
    dim3 dimGrid_pool2(batch, S2_channel);
    cuda_pool<<<dimGrid_pool2, dimBlock_pool2>>>(d_C2_2_feature_map, d_S2_feature_map);

    //////////BLOCK 3/////////////////////////////////
    // TODO: Implement pad
    dim3 dimBlock_pad3_1(S2_size, S2_size);
    dim3 dimGrid_pad3_1(batch, S2_channel);
    cuda_pad<<<dimGrid_pad3_1, dimBlock_pad3_1>>>(d_S2_feature_map, d_S2_feature_map_padded, conv3_1_padding_size);

    // TODO: Implement conv3_1
    dim3 dimBlock_conv3_1(C3_1_size, C3_1_size);
    dim3 dimGrid_conv3_1(batch, C3_1_channel);
    cuda_conv<<<dimGrid_conv3_1, dimBlock_conv3_1>>>(d_S2_feature_map_padded, d_C3_1_feature_map, d_conv3_1_weight, d_conv3_1_bias,
                                                        conv3_1_in_channel, conv3_1_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu3_1(C3_1_size, C3_1_size);
    dim3 dimGrid_relu3_1(batch, C3_1_channel);
    cuda_relu<<<dimGrid_relu3_1, dimBlock_relu3_1>>>(d_C3_1_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad3_2(C3_1_size, C3_1_size);
    dim3 dimGrid_pad3_2(batch, C3_1_channel);
    cuda_pad<<<dimGrid_pad3_2, dimBlock_pad3_2>>>(d_C3_1_feature_map, d_C3_1_feature_map_padded, conv3_2_padding_size);

    // TODO: Implement conv3_2
    dim3 dimBlock_conv3_2(C3_2_size, C3_2_size);
    dim3 dimGrid_conv3_2(batch, C3_2_channel);
    cuda_conv<<<dimGrid_conv3_2, dimBlock_conv3_2>>>(d_C3_1_feature_map_padded, d_C3_2_feature_map, d_conv3_2_weight, d_conv3_2_bias,
                                                        conv3_2_in_channel, conv3_2_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu3_2(C3_2_size, C3_2_size);
    dim3 dimGrid_relu3_2(batch, C3_2_channel);
    cuda_relu<<<dimGrid_relu3_2, dimBlock_relu3_2>>>(d_C3_2_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad3_3(C3_2_size, C3_2_size);
    dim3 dimGrid_pad3_3(batch, C3_2_channel);
    cuda_pad<<<dimGrid_pad3_3, dimBlock_pad3_3>>>(d_C3_2_feature_map, d_C3_2_feature_map_padded, conv3_3_padding_size);

    // TODO: Implement conv3_3
    dim3 dimBlock_conv3_3(C3_3_size, C3_3_size);
    dim3 dimGrid_conv3_3(batch, C3_3_channel);
    cuda_conv<<<dimGrid_conv3_3, dimBlock_conv3_3>>>(d_C3_2_feature_map_padded, d_C3_3_feature_map, d_conv3_3_weight, d_conv3_3_bias,
                                                        conv3_3_in_channel, conv3_3_kernel_size);
    // TODO: Implement relu
    dim3 dimBlock_relu3_3(C3_3_size, C3_3_size);
    dim3 dimGrid_relu3_3(batch, C3_3_channel);
    cuda_relu<<<dimGrid_relu3_3, dimBlock_relu3_3>>>(d_C3_3_feature_map);

    // TODO: Implement pool
    dim3 dimBlock_pool3(S3_size, S3_size);
    dim3 dimGrid_pool3(batch, S3_channel);
    cuda_pool<<<dimGrid_pool3, dimBlock_pool3>>>(d_C3_3_feature_map, d_S3_feature_map);

    //////////BLOCK 4/////////////////////////////////
    // TODO: Implement pad
    dim3 dimBlock_pad4_1(S3_size, S3_size);
    dim3 dimGrid_pad4_1(batch, S3_channel);
    cuda_pad<<<dimGrid_pad4_1, dimBlock_pad4_1>>>(d_S3_feature_map, d_S3_feature_map_padded, conv4_1_padding_size);

    // TODO: Implement conv4_1
    dim3 dimBlock_conv4_1(C4_1_size, C4_1_size);
    dim3 dimGrid_conv4_1(batch, C4_1_channel);
    cuda_conv<<<dimGrid_conv4_1, dimBlock_conv4_1>>>(d_S3_feature_map_padded, d_C4_1_feature_map, d_conv4_1_weight, d_conv4_1_bias,
                                                        conv4_1_in_channel, conv4_1_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu4_1(C4_1_size, C4_1_size);
    dim3 dimGrid_relu4_1(batch, C4_1_channel);
    cuda_relu<<<dimGrid_relu4_1, dimBlock_relu4_1>>>(d_C4_1_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad4_2(C4_1_size, C4_1_size);
    dim3 dimGrid_pad4_2(batch, C4_1_channel);
    cuda_pad<<<dimGrid_pad4_2, dimBlock_pad4_2>>>(d_C4_1_feature_map, d_C4_1_feature_map_padded, conv4_2_padding_size);

    // TODO: Implement conv4_2
    dim3 dimBlock_conv4_2(C4_2_size, C4_2_size);
    dim3 dimGrid_conv4_2(batch, C4_2_channel);
    cuda_conv<<<dimGrid_conv4_2, dimBlock_conv4_2>>>(d_C4_1_feature_map_padded, d_C4_2_feature_map, d_conv4_2_weight, d_conv4_2_bias,
                                                        conv4_2_in_channel, conv4_2_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu4_2(C4_2_size, C4_2_size);
    dim3 dimGrid_relu4_2(batch, C4_2_channel);
    cuda_relu<<<dimGrid_relu4_2, dimBlock_relu4_2>>>(d_C4_2_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad4_3(C4_2_size, C4_2_size);
    dim3 dimGrid_pad4_3(batch, C4_2_channel);
    cuda_pad<<<dimGrid_pad4_3, dimBlock_pad4_3>>>(d_C4_2_feature_map, d_C4_2_feature_map_padded, conv4_3_padding_size);

    // TODO: Implement conv4_3
    dim3 dimBlock_conv4_3(C4_3_size, C4_3_size);
    dim3 dimGrid_conv4_3(batch, C4_3_channel);
    cuda_conv<<<dimGrid_conv4_3, dimBlock_conv4_3>>>(d_C4_2_feature_map_padded, d_C4_3_feature_map, d_conv4_3_weight, d_conv4_3_bias,
                                                        conv4_3_in_channel, conv4_3_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu4_3(C4_3_size, C4_3_size);
    dim3 dimGrid_relu4_3(batch, C4_3_channel);
    cuda_relu<<<dimGrid_relu4_3, dimBlock_relu4_3>>>(d_C4_3_feature_map);

    // TODO: Implement pool
    dim3 dimBlock_pool4(S4_size, S4_size);
    dim3 dimGrid_pool4(batch, S4_channel);
    cuda_pool<<<dimGrid_pool4, dimBlock_pool4>>>(d_C4_3_feature_map, d_S4_feature_map);

    //////////BLOCK 5/////////////////////////////////
    // TODO: Implement pad
    dim3 dimBlock_pad5_1(S4_size, S4_size);
    dim3 dimGrid_pad5_1(batch, S4_channel);
    cuda_pad<<<dimGrid_pad5_1, dimBlock_pad5_1>>>(d_S4_feature_map, d_S4_feature_map_padded, conv5_1_padding_size);

    // TODO: Implement conv5_1
    dim3 dimBlock_conv5_1(C5_1_channel, 1);
    dim3 dimGrid_conv5_1(batch, 1);
    cuda_conv_b5<<<dimGrid_conv5_1, dimBlock_conv5_1>>>(d_S4_feature_map_padded, d_C5_1_feature_map, d_conv5_1_weight, d_conv5_1_bias,
                                                        C5_1_size, conv5_1_in_channel, conv5_1_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu5_1(C5_1_size, C5_1_size);
    dim3 dimGrid_relu5_1(batch, C5_1_channel);
    cuda_relu<<<dimGrid_relu5_1, dimBlock_relu5_1>>>(d_C5_1_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad5_2(C5_1_size, C5_1_size);
    dim3 dimGrid_pad5_2(batch, C5_1_channel);
    cuda_pad<<<dimGrid_pad5_2, dimBlock_pad5_2>>>(d_C5_1_feature_map, d_C5_1_feature_map_padded, conv5_2_padding_size);

    // TODO: Implement conv5_2
    dim3 dimBlock_conv5_2(C5_2_channel, 1);
    dim3 dimGrid_conv5_2(batch, 1);
    cuda_conv_b5<<<dimGrid_conv5_2, dimBlock_conv5_2>>>(d_C5_1_feature_map_padded, d_C5_2_feature_map, d_conv5_2_weight, d_conv5_2_bias,
                                                        C5_2_size, conv5_2_in_channel, conv5_2_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu5_2(C5_2_size, C5_2_size);
    dim3 dimGrid_relu5_2(batch, C5_2_channel);
    cuda_relu<<<dimGrid_relu5_2, dimBlock_relu5_2>>>(d_C5_2_feature_map);

    // TODO: Implement pad
    dim3 dimBlock_pad5_3(C5_2_size, C5_2_size);
    dim3 dimGrid_pad5_3(batch, C5_2_channel);
    cuda_pad<<<dimGrid_pad5_3, dimBlock_pad5_3>>>(d_C5_2_feature_map, d_C5_2_feature_map_padded, conv5_3_padding_size);

    // TODO: Implement conv5_3
    dim3 dimBlock_conv5_3(C5_3_channel, 1);
    dim3 dimGrid_conv5_3(batch, 1);
    cuda_conv_b5<<<dimGrid_conv5_3, dimBlock_conv5_3>>>(d_C5_2_feature_map_padded, d_C5_3_feature_map, d_conv5_3_weight, d_conv5_3_bias,
                                                        C5_3_size, conv5_3_in_channel, conv5_3_kernel_size);

    // TODO: Implement relu
    dim3 dimBlock_relu5_3(C5_3_size, C5_3_size);
    dim3 dimGrid_relu5_3(batch, C5_3_channel);
    cuda_relu<<<dimGrid_relu5_3, dimBlock_relu5_3>>>(d_C5_3_feature_map);

    // TODO: Implement pool
    dim3 dimBlock_pool5(S5_size, S5_size);
    dim3 dimGrid_pool5(batch, S5_channel);
    cuda_pool<<<dimGrid_pool5, dimBlock_pool5>>>(d_C5_3_feature_map, d_S5_feature_map);
    
    ////////////////////////////////////////////////////
    // TODO: Implement fc1
    dim3 dimBlock_fc(fc1_out_channel, 1);
    dim3 dimGrid_fc(batch, 1);
    cuda_fc<<<dimGrid_fc, dimBlock_fc>>>(d_S5_feature_map, d_output, d_fc1_weight, d_fc1_bias, fc1_in_channel);

    // TODO: Implement relu
    /* NOTE: unless you want to make a major change to this class structure, 
    *  you need to write your output to the device memory d_output 
    *  so that classify() can handle the rest.
    */
}

void vgg16_cuda::prepare_device_memory(uint8_t* image) {
  // Alloc Model Parameters

  //////////BLOCK 1/////////////////////////////////
  hipMalloc((void**)&d_conv1_1_weight,
             sizeof(float) * conv1_1_in_channel * conv1_1_out_channel *
                 conv1_1_kernel_size * conv1_1_kernel_size);
  hipMalloc((void**)&d_conv1_1_bias, sizeof(float) * conv1_1_out_channel);
  hipMalloc((void**)&d_conv1_2_weight,
             sizeof(float) * conv1_2_in_channel * conv1_2_out_channel *
                 conv1_2_kernel_size * conv1_2_kernel_size);
  hipMalloc((void**)&d_conv1_2_bias, sizeof(float) * conv1_2_out_channel);

  //////////BLOCK 2/////////////////////////////////
  hipMalloc((void**)&d_conv2_1_weight,
             sizeof(float) * conv2_1_in_channel * conv2_1_out_channel *
                 conv2_1_kernel_size * conv2_1_kernel_size);
  hipMalloc((void**)&d_conv2_1_bias, sizeof(float) * conv2_1_out_channel);
  hipMalloc((void**)&d_conv2_2_weight,
             sizeof(float) * conv2_2_in_channel * conv2_2_out_channel *
                 conv2_2_kernel_size * conv2_2_kernel_size);
  hipMalloc((void**)&d_conv2_2_bias, sizeof(float) * conv2_2_out_channel);

  //////////BLOCK 3/////////////////////////////////
  hipMalloc((void**)&d_conv3_1_weight,
             sizeof(float) * conv3_1_in_channel * conv3_1_out_channel *
                 conv3_1_kernel_size * conv3_1_kernel_size);
  hipMalloc((void**)&d_conv3_1_bias, sizeof(float) * conv3_1_out_channel);
  hipMalloc((void**)&d_conv3_2_weight,
             sizeof(float) * conv3_2_in_channel * conv3_2_out_channel *
                 conv3_2_kernel_size * conv3_2_kernel_size);
  hipMalloc((void**)&d_conv3_2_bias, sizeof(float) * conv3_2_out_channel);
  hipMalloc((void**)&d_conv3_3_weight,
             sizeof(float) * conv3_3_in_channel * conv3_3_out_channel *
                 conv3_3_kernel_size * conv3_3_kernel_size);
  hipMalloc((void**)&d_conv3_3_bias, sizeof(float) * conv3_3_out_channel);

  //////////BLOCK 4/////////////////////////////////
  hipMalloc((void**)&d_conv4_1_weight,
             sizeof(float) * conv4_1_in_channel * conv4_1_out_channel *
                 conv4_1_kernel_size * conv4_1_kernel_size);
  hipMalloc((void**)&d_conv4_1_bias, sizeof(float) * conv4_1_out_channel);
  hipMalloc((void**)&d_conv4_2_weight,
             sizeof(float) * conv4_2_in_channel * conv4_2_out_channel *
                 conv4_2_kernel_size * conv4_2_kernel_size);
  hipMalloc((void**)&d_conv4_2_bias, sizeof(float) * conv4_2_out_channel);
  hipMalloc((void**)&d_conv4_3_weight,
             sizeof(float) * conv4_3_in_channel * conv4_3_out_channel *
                 conv4_3_kernel_size * conv4_3_kernel_size);
  hipMalloc((void**)&d_conv4_3_bias, sizeof(float) * conv4_3_out_channel);

  //////////BLOCK 5/////////////////////////////////
  hipMalloc((void**)&d_conv5_1_weight,
             sizeof(float) * conv5_1_in_channel * conv5_1_out_channel *
                 conv5_1_kernel_size * conv5_1_kernel_size);
  hipMalloc((void**)&d_conv5_1_bias, sizeof(float) * conv5_1_out_channel);
  hipMalloc((void**)&d_conv5_2_weight,
             sizeof(float) * conv5_2_in_channel * conv5_2_out_channel *
                 conv5_2_kernel_size * conv5_2_kernel_size);
  hipMalloc((void**)&d_conv5_2_bias, sizeof(float) * conv5_2_out_channel);
  hipMalloc((void**)&d_conv5_3_weight,
             sizeof(float) * conv5_3_in_channel * conv5_3_out_channel *
                 conv5_3_kernel_size * conv5_3_kernel_size);
  hipMalloc((void**)&d_conv5_3_bias, sizeof(float) * conv5_3_out_channel);

  //////////FC 1////////////////////////////////////
  hipMalloc((void**)&d_fc1_weight,
             sizeof(float) * fc1_in_channel * fc1_out_channel);
  hipMalloc((void**)&d_fc1_bias, sizeof(float) * fc1_out_channel);

  // Alloc Activations
  hipMalloc((void**)&d_image,
             sizeof(uint8_t) * batch * input_size * input_size * input_channel);
  hipMalloc((void**)&d_input,
             sizeof(float) * batch * input_channel * input_size * input_size);

  //////////BLOCK 1/////////////////////////////////
  hipMalloc((void**)&d_input_padded,
             sizeof(float) * batch * input_channel * (input_size+2*conv1_1_padding_size) * (input_size+2*conv1_1_padding_size));
  hipMalloc((void**)&d_C1_1_feature_map,
             sizeof(float) * batch * C1_1_channel * C1_1_size * C1_1_size);
  hipMalloc((void**)&d_C1_1_feature_map_padded,
             sizeof(float) * batch * C1_1_channel * (C1_1_size+2*conv1_2_padding_size) * (C1_1_size+2*conv1_2_padding_size));
  hipMalloc((void**)&d_C1_2_feature_map,
             sizeof(float) * batch * C1_2_channel * C1_2_size * C1_2_size);
  hipMalloc((void**)&d_S1_feature_map,
             sizeof(float) * batch * S1_channel * S1_size * S1_size);

  //////////BLOCK 2/////////////////////////////////
  hipMalloc((void**)&d_S1_feature_map_padded,
             sizeof(float) * batch * S1_channel * (S1_size+2*conv2_1_padding_size) * (S1_size+2*conv2_1_padding_size));
  hipMalloc((void**)&d_C2_1_feature_map,
             sizeof(float) * batch * C2_1_channel * C2_1_size * C2_1_size);
  hipMalloc((void**)&d_C2_1_feature_map_padded,
             sizeof(float) * batch * C2_1_channel * (C2_1_size+2*conv2_2_padding_size) * (C2_1_size+2*conv2_2_padding_size));
  hipMalloc((void**)&d_C2_2_feature_map,
             sizeof(float) * batch * C2_2_channel * C2_2_size * C2_2_size);
  hipMalloc((void**)&d_S2_feature_map,
             sizeof(float) * batch * S2_channel * S2_size * S2_size);

  //////////BLOCK 3/////////////////////////////////
  hipMalloc((void**)&d_S2_feature_map_padded,
             sizeof(float) * batch * S2_channel * (S2_size+2*conv3_1_padding_size) * (S2_size+2*conv3_1_padding_size));
  hipMalloc((void**)&d_C3_1_feature_map,
             sizeof(float) * batch * C3_1_channel * C3_1_size * C3_1_size);
  hipMalloc((void**)&d_C3_1_feature_map_padded,
             sizeof(float) * batch * C3_1_channel * (C3_1_size+2*conv3_2_padding_size) * (C3_1_size+2*conv3_2_padding_size));
  hipMalloc((void**)&d_C3_2_feature_map,
             sizeof(float) * batch * C3_2_channel * C3_2_size * C3_2_size);
  hipMalloc((void**)&d_C3_2_feature_map_padded,
             sizeof(float) * batch * C3_2_channel * (C3_2_size+2*conv3_3_padding_size) * (C3_2_size+2*conv3_3_padding_size));
  hipMalloc((void**)&d_C3_3_feature_map,
             sizeof(float) * batch * C3_3_channel * C3_3_size * C3_3_size);
  hipMalloc((void**)&d_S3_feature_map,
             sizeof(float) * batch * S3_channel * S3_size * S3_size);

  //////////BLOCK 4/////////////////////////////////
  hipMalloc((void**)&d_S3_feature_map_padded,
             sizeof(float) * batch * S3_channel * (S3_size+2*conv4_1_padding_size) * (S3_size+2*conv4_1_padding_size));
  hipMalloc((void**)&d_C4_1_feature_map,
             sizeof(float) * batch * C4_1_channel * C4_1_size * C4_1_size);
  hipMalloc((void**)&d_C4_1_feature_map_padded,
             sizeof(float) * batch * C4_1_channel * (C4_1_size+2*conv4_2_padding_size) * (C4_1_size+2*conv4_2_padding_size));
  hipMalloc((void**)&d_C4_2_feature_map,
             sizeof(float) * batch * C4_2_channel * C4_2_size * C4_2_size);
  hipMalloc((void**)&d_C4_2_feature_map_padded,
             sizeof(float) * batch * C4_2_channel * (C4_2_size+2*conv4_3_padding_size) * (C4_2_size+2*conv4_3_padding_size));
  hipMalloc((void**)&d_C4_3_feature_map,
             sizeof(float) * batch * C4_3_channel * C4_3_size * C4_3_size);
  hipMalloc((void**)&d_S4_feature_map,
             sizeof(float) * batch * S4_channel * S4_size * S4_size);

  //////////BLOCK 5/////////////////////////////////
  hipMalloc((void**)&d_S4_feature_map_padded,
             sizeof(float) * batch * S4_channel * (S4_size+2*conv5_1_padding_size) * (S4_size+2*conv5_1_padding_size));
  hipMalloc((void**)&d_C5_1_feature_map,
             sizeof(float) * batch * C5_1_channel * C5_1_size * C5_1_size);
  hipMalloc((void**)&d_C5_1_feature_map_padded,
             sizeof(float) * batch * C5_1_channel * (C5_1_size+2*conv5_2_padding_size) * (C5_1_size+2*conv5_2_padding_size));
  hipMalloc((void**)&d_C5_2_feature_map,
             sizeof(float) * batch * C5_2_channel * C5_2_size * C5_2_size);
  hipMalloc((void**)&d_C5_2_feature_map_padded,
             sizeof(float) * batch * C5_2_channel * (C5_2_size+2*conv5_3_padding_size) * (C5_2_size+2*conv5_3_padding_size));
  hipMalloc((void**)&d_C5_3_feature_map,
             sizeof(float) * batch * C5_3_channel * C5_3_size * C5_3_size);
  hipMalloc((void**)&d_S5_feature_map,
             sizeof(float) * batch * S5_channel * S5_size * S5_size);


  hipMalloc((void**)&d_output, sizeof(float) * batch * output_size);

  // Copy Parameters
  //////////BLOCK 1/////////////////////////////////
  hipMemcpy(d_conv1_1_weight, conv1_1_weight,
             sizeof(float) * conv1_1_in_channel * conv1_1_out_channel *
                 conv1_1_kernel_size * conv1_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_1_bias, conv1_1_bias, sizeof(float) * conv1_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_2_weight, conv1_2_weight,
              sizeof(float) * conv1_2_in_channel * conv1_2_out_channel *
                  conv1_2_kernel_size * conv1_2_kernel_size,
              hipMemcpyHostToDevice);
   hipMemcpy(d_conv1_2_bias, conv1_2_bias, sizeof(float) * conv1_2_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 2/////////////////////////////////
  hipMemcpy(d_conv2_1_weight, conv2_1_weight,
             sizeof(float) * conv2_1_in_channel * conv2_1_out_channel *
                 conv2_1_kernel_size * conv2_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_1_bias, conv2_1_bias, sizeof(float) * conv2_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_2_weight, conv2_2_weight,
              sizeof(float) * conv2_2_in_channel * conv2_2_out_channel *
                  conv2_2_kernel_size * conv2_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_2_bias, conv2_2_bias, sizeof(float) * conv2_2_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 3/////////////////////////////////
  hipMemcpy(d_conv3_1_weight, conv3_1_weight,
             sizeof(float) * conv3_1_in_channel * conv3_1_out_channel *
                 conv3_1_kernel_size * conv3_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_1_bias, conv3_1_bias, sizeof(float) * conv3_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_2_weight, conv3_2_weight,
              sizeof(float) * conv3_2_in_channel * conv3_2_out_channel *
                  conv3_2_kernel_size * conv3_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_2_bias, conv3_2_bias, sizeof(float) * conv3_2_out_channel,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_3_weight, conv3_3_weight,
              sizeof(float) * conv3_3_in_channel * conv3_3_out_channel *
                  conv3_3_kernel_size * conv3_3_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv3_3_bias, conv3_3_bias, sizeof(float) * conv3_3_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 4/////////////////////////////////
  hipMemcpy(d_conv4_1_weight, conv4_1_weight,
             sizeof(float) * conv4_1_in_channel * conv4_1_out_channel *
                 conv4_1_kernel_size * conv4_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_1_bias, conv4_1_bias, sizeof(float) * conv4_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_2_weight, conv4_2_weight,
              sizeof(float) * conv4_2_in_channel * conv4_2_out_channel *
                  conv4_2_kernel_size * conv4_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_2_bias, conv4_2_bias, sizeof(float) * conv4_2_out_channel,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_3_weight, conv4_3_weight,
              sizeof(float) * conv4_3_in_channel * conv4_3_out_channel *
                  conv4_3_kernel_size * conv4_3_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv4_3_bias, conv4_3_bias, sizeof(float) * conv4_3_out_channel,
              hipMemcpyHostToDevice);

  //////////BLOCK 5/////////////////////////////////
  hipMemcpy(d_conv5_1_weight, conv5_1_weight,
             sizeof(float) * conv5_1_in_channel * conv5_1_out_channel *
                 conv5_1_kernel_size * conv5_1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_1_bias, conv5_1_bias, sizeof(float) * conv5_1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_2_weight, conv5_2_weight,
              sizeof(float) * conv5_2_in_channel * conv5_2_out_channel *
                  conv5_2_kernel_size * conv5_2_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_2_bias, conv5_2_bias, sizeof(float) * conv5_2_out_channel,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_3_weight, conv5_3_weight,
              sizeof(float) * conv5_3_in_channel * conv5_3_out_channel *
                  conv5_3_kernel_size * conv5_3_kernel_size,
              hipMemcpyHostToDevice);
  hipMemcpy(d_conv5_3_bias, conv5_3_bias, sizeof(float) * conv5_3_out_channel,
              hipMemcpyHostToDevice);


  hipMemcpy(d_fc1_weight, fc1_weight,
             sizeof(float) * fc1_in_channel * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_bias, fc1_bias, sizeof(float) * fc1_out_channel,
             hipMemcpyHostToDevice);

  // copy input image
  size_t image_size = batch * input_size * input_size * input_channel;
  hipMemcpy(d_image, image, image_size * sizeof(uint8_t),
             hipMemcpyHostToDevice);
}

void vgg16_cuda::classify(int* predict, int batch) {
  // read logits back to cpu
  hipMemcpy(output, d_output, sizeof(float) * output_size * batch,
             hipMemcpyDeviceToHost);
  // Softmax
  softmax(output, predict, batch, output_size);
}

vgg16_cuda::~vgg16_cuda() {
  hipFree(d_conv1_1_weight);   
  hipFree(d_conv1_2_weight);   
  hipFree(d_conv2_1_weight);   
  hipFree(d_conv2_2_weight);  
  hipFree(d_conv3_1_weight);   
  hipFree(d_conv3_2_weight);   
  hipFree(d_conv3_3_weight);   
  hipFree(d_conv4_1_weight);   
  hipFree(d_conv4_2_weight);   
  hipFree(d_conv4_3_weight); 
  hipFree(d_conv5_1_weight);   
  hipFree(d_conv5_2_weight);   
  hipFree(d_conv5_3_weight);   
 
  hipFree(d_conv1_1_bias);   
  hipFree(d_conv1_2_bias);   
  hipFree(d_conv2_1_bias);   
  hipFree(d_conv2_2_bias);  
  hipFree(d_conv3_1_bias);   
  hipFree(d_conv3_2_bias);   
  hipFree(d_conv3_3_bias);   
  hipFree(d_conv4_1_bias);   
  hipFree(d_conv4_2_bias);   
  hipFree(d_conv4_3_bias); 
  hipFree(d_conv5_1_bias);   
  hipFree(d_conv5_2_bias);   
  hipFree(d_conv5_3_bias);   
   
  hipFree(d_fc1_weight);     
  hipFree(d_fc1_bias);        

  hipFree(d_image);          
  hipFree(d_input); 

  hipFree(d_input_padded);          
  hipFree(d_C1_1_feature_map); 
  hipFree(d_C1_1_feature_map_padded); 
  hipFree(d_C1_2_feature_map); 
  hipFree(d_S1_feature_map); 

  hipFree(d_S1_feature_map_padded); 
  hipFree(d_C2_1_feature_map); 
  hipFree(d_C2_1_feature_map_padded); 
  hipFree(d_C2_2_feature_map); 
  hipFree(d_S2_feature_map); 

  hipFree(d_S2_feature_map_padded); 
  hipFree(d_C3_1_feature_map); 
  hipFree(d_C3_1_feature_map_padded); 
  hipFree(d_C3_2_feature_map); 
  hipFree(d_C3_2_feature_map_padded); 
  hipFree(d_C3_3_feature_map); 
  hipFree(d_S3_feature_map); 

  hipFree(d_S3_feature_map_padded); 
  hipFree(d_C4_1_feature_map); 
  hipFree(d_C4_1_feature_map_padded); 
  hipFree(d_C4_2_feature_map); 
  hipFree(d_C4_2_feature_map_padded); 
  hipFree(d_C4_3_feature_map); 
  hipFree(d_S4_feature_map); 

  hipFree(d_S4_feature_map_padded); 
  hipFree(d_C5_1_feature_map); 
  hipFree(d_C5_1_feature_map_padded); 
  hipFree(d_C5_2_feature_map); 
  hipFree(d_C5_2_feature_map_padded); 
  hipFree(d_C5_3_feature_map); 
  hipFree(d_S5_feature_map); 
 
  hipFree(d_output);       
  hipFree(d_predict_cuda);   
}
